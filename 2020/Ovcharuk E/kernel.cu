
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

hipError_t addWithCuda(int *a, unsigned int size);

__global__ void addKernel(int *a)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int theNum = a[i];

    int first = theNum / 100;
    int sec = (theNum - (first * 100)) / 10;
    int last = (theNum - (first * 100)) - (sec * 10);

    int sum = (first * first * first) + (sec * sec * sec) + (last * last * last);
    if (sum == theNum) {
        a[i] = 1;
    }
    else {
        a[i] = 0;
    }
}

int main()
{
    const int arraySize = 900;
    int seq[arraySize];
    int count = 100;

    for (int i = 0; i < 900; i++) {
        seq[i] = count++;
    }
    
    // find Armstrong numbers with GPU
    hipError_t cudaStatus = addWithCuda(seq, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    std::cout << "Armstrong numbers:" << std::endl;

    for (int i = 0; i < 900; i++) {
        if (seq[i] > 0) {
            std::cout << i + 100 << std::endl;
        }
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *a, unsigned int size)
{
    int *dev_a = 0;
    
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_a);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    
    return cudaStatus;
}
