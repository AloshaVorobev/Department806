
#include "hip/hip_runtime.h"

#include <ctime>
#include<stdio.h>
#include<stdlib.h>

__global__ void print_from_gpu() {
	printf("Cuda lab1 \n");
}

int main() {
	int n=10;

//GPU  
  hipEvent_t start, stop;
  float gpuTime =0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start,0);
  print_from_gpu<<<10,1>>>();
  hipEventRecord(stop,0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpuTime, start, stop);
  printf("time on GPU = %.4f ms\n", gpuTime);

	hipDeviceSynchronize();
  hipEventDestroy(start);
  hipEventDestroy(stop);

//CPU
  clock_t c_start, c_stop;  
  clock_t begin = clock();

	for (int i = 0; i < n; i++)
	{
		printf("Cuda lab1 \n");
	}

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;

	printf("time on CPU = %.4f ms\n", time_spent);


	return 0;
}
