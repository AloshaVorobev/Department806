#include "hip/hip_runtime.h"

#include <ctime>
#include <stdio.h>

__global__ void squareKernel(int* data, int N);

int main(int argc, char** argv)
{
	int* arr;
	int* h_data;
	int* d_data;
	//���������� ��������� + 1
	int n = 1000000;
	//����� ���������
	int sum = 0;
	int sumCPU = 0;

	//������ ��� �������� ������� ���������� �� GPU
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	//clock ��� �������� ������� ���������� �� CPU
	clock_t c_start, c_stop;

	// �������� page-locked ������ �� �����
	// ��� ������� ����� ����� ������������ �������� ��� ��������� ������������� �������� ��� ������ ������� ����� ������ � �����������.
	hipHostAlloc(&h_data, n * sizeof(int), hipHostMallocPortable);

	//cudaMemcpy(h_data, arr, n * sizeof(int), cudaMemcpyHostToDevice);

	// �������� ������ �� ����������
	hipMalloc(&d_data, n * sizeof(int));

	dim3 block(512);
	dim3 grid((n + block.x - 1) / block.x);

	//grid - ���������� ������
	//block - ������ �����
	squareKernel <<<grid, block>>>(d_data, n);

	hipEventRecord(g_start, 0);

	//�������� ������ � ���������� (d_data) �� ���� (h_data)
	hipMemcpy(h_data, d_data, n * sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float GPUelapsedTime;
	hipEventElapsedTime(&GPUelapsedTime, g_start, g_stop);

	for (int j = 0; j < n; j++)
	{
		sum = sum + h_data[j];
	}

	//CPU
	clock_t begin = clock();

	for (int i = 0; i < n; i++)
	{
		sumCPU += i * i;
	}

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;

	printf("Time in GPU: %lfms\n", GPUelapsedTime);
	printf("Time in CPU: %lfms\n", time_spent);
	printf("sum GPU for %d = %d\n", n-1, sum);
	printf("sum CPU for %d = %d\n", n-1, sumCPU);

	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);
	return 0;
}

__global__ void squareKernel(int* data, int N)
{
	//threadIdx � ����� ���� � �����
	//blockIdx � ����� �����, � ������� ��������� ����
	//blockDim � ������ �����

	//���������� ������ ���� ������ ����
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		data[i] = i * i;
	}
}