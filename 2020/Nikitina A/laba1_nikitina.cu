
#include "hip/hip_runtime.h"

#include <stdio.h>

#define n 10 //����� �������

//���� ����������� ���������� �� ������� ����� �����
__global__ void kernel(int* a, int* b, int* c)
{
	//���������� ������ ����
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//��������� ��������� �������������� ������ ���� ������
	c[idx] = a[idx] * b[idx];
}

int main(void)
{
	int numBytes = n * sizeof(int);
	int a[n], b[n], c[n];
	int* adev, * bdev, * cdev;

	//�������� ������ �� GPU
	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&bdev, numBytes);
	hipMalloc((void**)&cdev, numBytes);

	//������ �������
	for (int i = 0; i < n; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}

	//����������� ������� ������ �� ������ CPU � ������ GPU
	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

	//����� ���� � �������� ������������� �������
	kernel <<<n, 1 >>> (adev, bdev, cdev);

	//����������� ���������� � ������ CPU
	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);

	//����� ����������
	for (int idx = 0; idx < n; idx++)
	{
		printf("%d * %d = %d \n", a[idx], b[idx], c[idx]);
	}

	//���������� ���������� ������ GPU
	hipFree(adev);
	hipFree(bdev);
	hipFree(cdev);

	return 0;
}
