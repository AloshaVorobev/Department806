#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <chrono>
typedef std::chrono::high_resolution_clock Clock;

#define CSC(call) {                                 \
    hipError_t err = call;                         \
    if(err != hipSuccess) {                        \
        fprintf(stderr, "CUDA Error in file %s in line %d: %s.\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(0);                                    \
    }                                               \
} while(0)

#define N 65535


__global__ void add_kernel(int *a)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    for (int i = tid; i < N; i += offset)
        a[i] = i + 1;
}


int main(void)
{
	int a[N];
	int *dev_a;

	int threadsPerBlock = 1024;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));
	CSC(hipEventRecord(start, 0));

	CSC(hipMalloc((void**)&dev_a, N * sizeof(int)));

	add_kernel<<<blocksPerGrid, threadsPerBlock >>>(dev_a);

	CSC(hipMemcpy(a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost));

	CSC(hipDeviceSynchronize());
	CSC(hipEventRecord(stop, 0));
	CSC(hipEventSynchronize(stop));
	CSC(hipEventElapsedTime(&gpuTime, start, stop));

	//for (int i = 0; i < N; ++i)
	//	printf("%d\t", a[i]);

	printf("\n===================   GPU    ===================\n");
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	printf("DEVICE GPU compute time: %f milliseconds\n", gpuTime);

	CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(stop));
    CSC(hipFree(dev_a));
    free(a);

	// CPU
	double time_CPU;
    int a2[N];
    auto t1 = Clock::now();

	for (int i = 0; i < N; ++i)
		a2[i] = i + 1;

	//for (int i = 0; i < N; ++i)
		//printf("%d\t", a2[i]);

	auto t2 = Clock::now();
    time_CPU = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();

	printf("\n===================   CPU    ===================\n");
	printf("HOST CPU compute time: %f milliseconds\n", time_CPU);

    time_CPU = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
    printf("HOST CPU compute time: %f microseconds\n", time_CPU);

    free(a2);

	return 0;
}